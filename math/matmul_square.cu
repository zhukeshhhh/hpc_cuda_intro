#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

__global__ void matrixMulGpu(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // each thread's row
    int col = blockIdx.x * blockDim.x + threadIdx.x; // each thread's col

    int temp_sum = 0;
    if ((row < n) && (col < n)) {
        // iterate over row and down column
        for (int k = 0; k < n; k++) {
            temp_sum += a[row * n + k] * b[k * n + col];
        }
        // assign result
        c[row * n + col] = temp_sum;
    }
}

void matrixMulCpu(int *a, int *b, int *c, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) [
            int temp_sum = 0;
            for (int k = 0; k < )
        ]
    }
}

void init_matrix(int *m, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            m[i * n + j] = rand() % 100;
        }
    }
}

int main() {

    int n  = 1 << 10; // matrix size of 1024 (2^10)

    size_t bytes = n * n * sizeof(int);

    int *h_a, *h_b, *h_c; // host pointers

    // allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    //device pointers
    int *d_a, *d_b, *d_c;

    //allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // init matrices
    init_matrix(h_a, n);
    init_matrix(h_b, n);

    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);


    int BLOCK_SIZE = 16; // threads per one block
    int GRID_SIZE = (int)ceil(n / BLOCK_SIZE); // block in each dimension

    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    matrixMulGpu<<<grid, threads>>>(d_a, d_b, d_c, n); // launch kernel

    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy( h_b, d_b, bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            int idx = i * n + j;
            printf("c[%d][%d] = %d\n", i, j, h_c[idx]);
        }
}


    printf("SUCCESS!\n");

    return 0;
}