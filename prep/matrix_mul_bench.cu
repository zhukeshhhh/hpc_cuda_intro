
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

vector<int> makeTestSizes(int start, int end) {
    vector<int> sizes;
    for (int n = start; n <= end; n += 256)
        sizes.push_back(n);
    return sizes;
}

// naive kernel
__global__ void matMulNaive(const int* A, const int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        int tmp = 0;
        for (int k = 0; k < N; ++k)
            tmp += A[row * N + k] * B[k * N + col];
        C[row * N + col] = tmp;
    }
}

// tiled shared-memory kernel
const int TILE = 32;
__global__ void matMulTiled(const int* A, const int* B, int* C, int N) {
    __shared__ int sA[TILE][TILE];
    __shared__ int sB[TILE][TILE];

    int row = blockIdx.y * TILE + threadIdx.y;
    int col = blockIdx.x * TILE + threadIdx.x;
    int tmp = 0;

    int numTiles = (N + TILE - 1) / TILE;
    for (int t = 0; t < numTiles; ++t) {
        int aIdx = row * N + t * TILE + threadIdx.x;
        int bIdx = (t * TILE + threadIdx.y) * N + col;
        sA[threadIdx.y][threadIdx.x] = (row < N && (t * TILE + threadIdx.x) < N)
                                        ? A[aIdx] : 0;
        sB[threadIdx.y][threadIdx.x] = (col < N && (t * TILE + threadIdx.y) < N)
                                        ? B[bIdx] : 0;
        __syncthreads();

        for (int k = 0; k < TILE; ++k)
            tmp += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = tmp;
}

int main() {
    auto sizes = makeTestSizes(1024, 16384);
    std::ofstream csv("benchmark.csv");
    csv << "N,naive_ms,tiled_ms\n";

    const int numRuns = 10;

    // -------- Warmup GPU --------
    {
        int N = 512;
        size_t bytes = size_t(N) * N * sizeof(int);
        vector<int> hA(N * N, 1), hB(N * N, 1), hC(N * N);
        int *dA, *dB, *dC;
        hipMalloc(&dA, bytes);
        hipMalloc(&dB, bytes);
        hipMalloc(&dC, bytes);
        hipMemcpy(dA, hA.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(dB, hB.data(), bytes, hipMemcpyHostToDevice);
        dim3 threads(TILE, TILE);
        dim3 blocks((N + TILE - 1) / TILE, (N + TILE - 1) / TILE);
        matMulNaive<<<blocks, threads>>>(dA, dB, dC, N);
        matMulTiled<<<blocks, threads>>>(dA, dB, dC, N);
        hipDeviceSynchronize();
        hipFree(dA);
        hipFree(dB);
        hipFree(dC);
    }

    for (int N : sizes) {
        size_t bytes = size_t(N) * N * sizeof(int);

        // host data
        vector<int> hA(N * N), hB(N * N), hC(N * N);
        generate(hA.begin(), hA.end(), []() { return rand() % 100; });
        generate(hB.begin(), hB.end(), []() { return rand() % 100; });

        // device allocations
        int *dA, *dB, *dC;
        hipMalloc(&dA, bytes);
        hipMalloc(&dB, bytes);
        hipMalloc(&dC, bytes);
        hipMemcpy(dA, hA.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(dB, hB.data(), bytes, hipMemcpyHostToDevice);

        dim3 threads(TILE, TILE);
        dim3 blocks((N + TILE - 1) / TILE, (N + TILE - 1) / TILE);

        float total_naive = 0.0f, total_tiled = 0.0f;

        for (int run = 0; run < numRuns; ++run) {
            hipEvent_t start, stop;
            float ms;

            // naive
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            matMulNaive<<<blocks, threads>>>(dA, dB, dC, N);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&ms, start, stop);
            total_naive += ms;
            hipEventDestroy(start);
            hipEventDestroy(stop);

            // tiled
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            matMulTiled<<<blocks, threads>>>(dA, dB, dC, N);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&ms, start, stop);
            total_tiled += ms;
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        float avg_naive = total_naive / numRuns;
        float avg_tiled = total_tiled / numRuns;

        // record & report
        csv << N << "," << avg_naive << "," << avg_tiled << "\n";
        cout << "N = " << N << " | naive(avg) = " << avg_naive
             << " ms, tiled(avg) = " << avg_tiled << " ms\n";

        hipFree(dA);
        hipFree(dB);
        hipFree(dC);
    }

    csv.close();
    cout << "Done. Results in benchmark.csv\n";
    return 0;
}
